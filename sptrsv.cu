
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
//#include <hipsparse/hipsparse.h>
#include <stdio.h>

typedef struct {
    int row;
    int col;
    int val;
} Entry_t;


__global__ void helloCUDA() {
    printf("Hello, CUDA!\n");
}
__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    std::ifstream input("mawi_201512020330/mawi_201512020330.mtx");
    std::string line;
    // Get the first line
    getline(input, line);
    std::cout << line << std::endl;
    std::stringstream stream(line);
    std::string token;

    getline(stream, token, ' ');
    int nrows = std::stoi(token);
    getline(stream, token, ' ');
    int ncols = std::stoi(token);
    getline(stream, token, ' ');
    int nnz  = std::stoi(token);

    std::cout << "(" << nrows << "," << ncols << ")" << nnz <<std::endl;

//    int row, col;
//    int val;
//    int count = 0;

    // Start parsing the matrix
//    while(getline( input, line )){
//	std::stringstream ss(line);
//	getline(ss, token, ' ');
//	row = std::stoi(token);
//	getline(ss, token, ' ');
//	col = std::stoi(token);
//	getline(ss, token, ' ');
//	val = std::stoi(token);
//	Entry_t *ptr = new Entry_t;
//	ptr->row =row;
//	ptr->col = col;
//	ptr->val = val;
//	count++;
//	v.push_back(ptr);
//	if (row != col) {
//	    // also need to add in the transpose
//	    Entry_t *transpose = new Entry_t;
//	    transpose->row = col;
//	    transpose->col = row;
//	    transpose->val = val;
//	    v.push_back(transpose);
//	    count++;
//	}
//
//    }
//    std::cout << "Found " << count << " nnz entries" << std::endl; 
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    float *a, *b, *out;
    float *d_a, *d_out;

    int N = 10;

    a = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = i*1.0f; 
	b[i] = i*2.0f;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    vector_add<<<1,1>>>(out, d_a, b, N);
    hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++){
        std::cout << a[i] << "+" << b[i] << "=" << out[i] << std::endl;
    }
    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);

    std::cout << "End of program" <<std::endl;
    return 0;
}
