#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <cassert>
#include <sstream>
#include <stdio.h>
#include <algorithm>
#include "MatrixLoader.hpp"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int i = threadIdx.x + blockDim.x * threadIdx.y;
    if (i < n) {
        out[i] = a[i] + b[i]; 
    }
}

int main(){
    MatrixLoader *myld = new MatrixLoader("arc130/arc130.mtx", 3.959802e-31);
    myld->printConfigs();
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    int N = 10;

    a = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = i*1.0f; 
	b[i] = i*2.0f;
    }

    hipError_t result;
    // Allocate device memory for a
    result = hipMalloc((void**)&d_a, sizeof(float) * N);
    assert(result == hipSuccess);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    result = hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    assert(result == hipSuccess);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    vector_add<<<1,256>>>(d_out, d_a, d_b, N);
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++){
        std::cout << a[i] << "+" << b[i] << "=" << out[i] << std::endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);

    delete myld;
    std::cout << "End of program" <<std::endl;
    return 0;
}
