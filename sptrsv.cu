
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <cassert>
#include <sstream>
#include <stdio.h>

typedef struct {
    int row;
    int col;
    int val;
} Entry_t;

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int i = threadIdx.x + blockDim.x * threadIdx.y;
    if (i < n) {
        out[i] = a[i] + b[i]; 
    }
}

int main(){
    std::ifstream input("mawi_201512020330/mawi_201512020330.mtx");
    std::string line;
    // Get the first line
    getline(input, line);
    std::cout << line << std::endl;
    std::stringstream stream(line);
    std::string token;

    getline(stream, token, ' ');
    int nrows = std::stoi(token);
    getline(stream, token, ' ');
    int ncols = std::stoi(token);
    getline(stream, token, ' ');
    int nnz  = std::stoi(token);

    std::cout << "(" << nrows << "," << ncols << ")" << nnz <<std::endl;


    // Start parsing the matrix
//    while(getline( input, line )){
//	std::stringstream ss(line);
//	getline(ss, token, ' ');
//	row = std::stoi(token);
//	getline(ss, token, ' ');
//	col = std::stoi(token);
//	getline(ss, token, ' ');
//	val = std::stoi(token);
//	Entry_t *ptr = new Entry_t;
//	ptr->row =row;
//	ptr->col = col;
//	ptr->val = val;
//	count++;
//	v.push_back(ptr);
//	if (row != col) {
//	    // also need to add in the transpose
//	    Entry_t *transpose = new Entry_t;
//	    transpose->row = col;
//	    transpose->col = row;
//	    transpose->val = val;
//	    v.push_back(transpose);
//	    count++;
//	}
//
//    }
//    std::cout << "Found " << count << " nnz entries" << std::endl; 
    float *a, *b, *out;
    float *c, *d;
    float *d_a, *d_b, *d_out;

    int N = 10;

    a = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    c = (float*)malloc(sizeof(float) * N);
    d   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = i*1.0f; 
	b[i] = i*2.0f;
    }
    for(int i = 0; i < N; i++){
        printf("a=%f,b=%f\n", a[i], b[i]);
    }

    hipError_t result;
    // Allocate device memory for a
    result = hipMalloc((void**)&d_a, sizeof(float) * N);
    assert(result == hipSuccess);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    result = hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    assert(result == hipSuccess);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(c, d_a, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(d, d_b, sizeof(float) * N, hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++){
        printf("c=%f,d=%f\n", c[i], d[i]);
    }
    vector_add<<<1,256>>>(d_out, d_a, d_b, N);
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++){
        std::cout << a[i] << "+" << b[i] << "=" << out[i] << std::endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(c);
    free(d);
    free(out);

    std::cout << "End of program" <<std::endl;
    return 0;
}
