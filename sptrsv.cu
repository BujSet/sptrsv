#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <cassert>
#include <sstream>
#include <stdio.h>
#include <algorithm>
#include "MatrixLoader.hpp"

bool compareByRow(const Entry_t *elem1, const Entry_t *elem2 ) {
    if (elem1->row == elem2->row) {
        return elem1->col < elem2->col;
    }
    return elem1->row < elem2->row;
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int i = threadIdx.x + blockDim.x * threadIdx.y;
    if (i < n) {
        out[i] = a[i] + b[i]; 
    }
}

int main(){
//    std::ifstream input("mawi_201512020330/mawi_201512020330.mtx");
    std::ifstream input("arc130/arc130.mtx");
    std::string line;
    // Get the first line
    getline(input, line);
    std::cout << line << std::endl;
    std::stringstream stream(line);
    std::string token;

    getline(stream, token, ' ');
    int nrows = std::stoi(token);
    getline(stream, token, ' ');
    int ncols = std::stoi(token);
    getline(stream, token, ' ');
    int nnz  = std::stoi(token);

    std::cout << "(" << nrows << "," << ncols << ")" << nnz <<std::endl;

    MatrixLoader *myld = new MatrixLoader("arc130/arc130.mtx", 3.959802e-31);


    int row, col;
    float val;
    int count = 0;
    std::vector<Entry_t*> v;
    // Start parsing the matrix
    while(getline( input, line )){
	std::stringstream ss(line);
	getline(ss, token, ' ');
	row = std::stoi(token);
	getline(ss, token, ' ');
	col = std::stoi(token);
	getline(ss, token, ' ');
	val = std::stof(token);
	if (std::abs(val) < 3.959802e-31) {
	    printf("Skipping value =%f\n", val);
            continue;
	}
	Entry_t *ptr = new Entry_t;
	ptr->row =row;
	ptr->col = col;
	ptr->val = val;
	count++;
	v.push_back(ptr);
    }
    std::sort(v.begin(), v.end(), &compareByRow);
    std::cout << "Found " << count << " nnz entries" << std::endl; 
    for(Entry_t *ptr : v) {
        std::cout << "(" << ptr->row << "," << ptr->col << ")=" << ptr->val << std::endl;
    }
    int *csrRowPtrs = (int*)malloc(sizeof(int)*nnz);
    int *csrColIdxs = (int*)malloc(sizeof(int)*nnz);
    float *csrVals = (float*)malloc(sizeof(float)*nnz);
    float *a, *b, *out;
    float *c, *d;
    float *d_a, *d_b, *d_out;

    int N = 10;

    a = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    c = (float*)malloc(sizeof(float) * N);
    d   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = i*1.0f; 
	b[i] = i*2.0f;
    }
    for(int i = 0; i < N; i++){
        printf("a=%f,b=%f\n", a[i], b[i]);
    }

    hipError_t result;
    // Allocate device memory for a
    result = hipMalloc((void**)&d_a, sizeof(float) * N);
    assert(result == hipSuccess);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    result = hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    assert(result == hipSuccess);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(c, d_a, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(d, d_b, sizeof(float) * N, hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++){
        printf("c=%f,d=%f\n", c[i], d[i]);
    }
    vector_add<<<1,256>>>(d_out, d_a, d_b, N);
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++){
        std::cout << a[i] << "+" << b[i] << "=" << out[i] << std::endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(c);
    free(d);
    free(csrRowPtrs);
    free(csrColIdxs);
    free(csrVals);
    free(out);

    for (Entry_t *ptr : v) {
        delete ptr;
    }
    delete myld;
    std::cout << "End of program" <<std::endl;
    return 0;
}
