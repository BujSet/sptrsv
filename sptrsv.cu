
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
//#include <hipsparse/hipsparse.h>
#include <stdio.h>

typedef struct {
    int row;
    int col;
    int val;
} Entry_t;


__global__ void helloCUDA() {
    printf("Hello, CUDA!\n");
}

int main(){
    std::ifstream input("mawi_201512020330/mawi_201512020330.mtx");
    std::string line;
    // Get the first line
    getline(input, line);
    std::cout << line << std::endl;
    std::stringstream stream(line);
    std::string token;

    getline(stream, token, ' ');
    int nrows = std::stoi(token);
    getline(stream, token, ' ');
    int ncols = std::stoi(token);
    getline(stream, token, ' ');
    int nnz  = std::stoi(token);

    std::cout << "(" << nrows << "," << ncols << ")" << nnz <<std::endl;

//    int row, col;
//    int val;
//    int count = 0;

    // Start parsing the matrix
//    while(getline( input, line )){
//	std::stringstream ss(line);
//	getline(ss, token, ' ');
//	row = std::stoi(token);
//	getline(ss, token, ' ');
//	col = std::stoi(token);
//	getline(ss, token, ' ');
//	val = std::stoi(token);
//	Entry_t *ptr = new Entry_t;
//	ptr->row =row;
//	ptr->col = col;
//	ptr->val = val;
//	count++;
//	v.push_back(ptr);
//	if (row != col) {
//	    // also need to add in the transpose
//	    Entry_t *transpose = new Entry_t;
//	    transpose->row = col;
//	    transpose->col = row;
//	    transpose->val = val;
//	    v.push_back(transpose);
//	    count++;
//	}
//
//    }
//    std::cout << "Found " << count << " nnz entries" << std::endl; 
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
